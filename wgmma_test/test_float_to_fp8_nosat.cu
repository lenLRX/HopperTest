
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h>
#include <math.h>

#include <hip/hip_fp8.h>

__global__ void test_main(float* input, uint16_t *output, __hip_saturation_t sat_opt) {
  //float a = NAN;
  //float b = INFINITY;
  //float o = 1000000;
  float a = input[0];
  float b = input[1];
  float o = input[2];

  uint16_t c;
  uint16_t d;
  uint16_t e;
  uint16_t f;
  uint16_t g;
  uint16_t h;

  c = __hip_cvt_float_to_fp8(a, sat_opt, __HIP_E4M3_FNUZ);
  d = __hip_cvt_float_to_fp8(b, sat_opt, __HIP_E4M3_FNUZ);
  e = __hip_cvt_float_to_fp8(o, sat_opt, __HIP_E4M3_FNUZ);

  f = __hip_cvt_float_to_fp8(a, sat_opt, __HIP_E5M2_FNUZ);
  g = __hip_cvt_float_to_fp8(b, sat_opt, __HIP_E5M2_FNUZ);
  h = __hip_cvt_float_to_fp8(o, sat_opt, __HIP_E5M2_FNUZ);



  output[0] = c;
  output[1] = d;
  output[2] = e;

  output[3] = f;
  output[4] = g;
  output[5] = h;


}

int main() {
    float* input_host = new float[3];
    input_host[0] = NAN;
    input_host[1] = INFINITY;
    input_host[2] = 1000000;

    float* input_device;
    hipMalloc(&input_device, 3 * sizeof(float));
    hipMemcpy(input_device, input_host, 3*sizeof(float),
                       hipMemcpyHostToDevice);

    uint16_t* output_host = new uint16_t[6];
    uint16_t* output_device;
    hipMalloc(&output_device, 6 * sizeof(uint16_t));

    test_main<<<1,1>>>(input_device, output_device, __HIP_NOSAT);

    hipDeviceSynchronize();
    hipMemcpy(output_host, output_device, 6*sizeof(uint16_t),
                       hipMemcpyDeviceToHost);
    std::cout << "e4m3 nan cast nosat: " << (output_host[0]&0xff) << "\n";
    std::cout << "e4m3 inf cast nosat: " << (output_host[1]&0xff) << "\n";
    std::cout << "e4m3 ovf cast nosat: " << (output_host[2]&0xff) << "\n";
    std::cout << "e5m2 nan cast nosat: " << (output_host[3]&0xff) << "\n";
    std::cout << "e5m2 inf cast nosat: " << (output_host[4]&0xff) << "\n";
    std::cout << "e5m2 ovf cast nosat: " << (output_host[5]&0xff) << "\n";


    test_main<<<1,1>>>(input_device, output_device, __HIP_SATFINITE);

    hipDeviceSynchronize();
    hipMemcpy(output_host, output_device, 6*sizeof(uint16_t),
                       hipMemcpyDeviceToHost);
    std::cout << "e4m3 nan cast satfinite: " << (output_host[0]&0xff) << "\n";
    std::cout << "e4m3 inf cast satfinite: " << (output_host[1]&0xff) << "\n";
    std::cout << "e4m3 ovf cast satfinite: " << (output_host[2]&0xff) << "\n";
    std::cout << "e5m2 nan cast satfinite: " << (output_host[3]&0xff) << "\n";
    std::cout << "e5m2 inf cast satfinite: " << (output_host[4]&0xff) << "\n";
    std::cout << "e5m2 ovf cast satfinite: " << (output_host[5]&0xff) << "\n";




}
