#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_fp16.h>
#include <fstream>
#include <iostream>

#include "gmma_desc.h"

// code from
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

static const int M = 64;
static const int N = 256;
static const int K = 16;

// I didn't check result since it is just a throughput test
__global__ void wgmma_m64n256k16_throughput_fmix_pingpong_test(
    float *gm_d, __half2 *gm_a, __half2 *gm_b, float *gm_c, uint8_t sm_layout,
    uint64_t repeat_time) {
  int warp_group_id = threadIdx.x / 32 / 4;
  int tid_in_wg = threadIdx.x % 128;

  int mat_a_size = M * K;
  int mat_b_size = N * K;

  int wg_shared_mem_size =
      mat_a_size * sizeof(__half) + mat_b_size * sizeof(__half);

  extern __shared__ char sm_buff[];
  float RegD[128];

  uint64_t *barrier_ptr = (uint64_t *)(sm_buff + wg_shared_mem_size * 2);
  uint32_t bar_sm_addr =
      static_cast<uint32_t>(__cvta_generic_to_shared(barrier_ptr));

  if (threadIdx.x == 0) {
    int arrive_count = 1;

    asm volatile("{\n\t"
                 "mbarrier.init.shared.b64 [%1], %0; \n"
                 "}"
                 :
                 : "r"(arrive_count), "r"(bar_sm_addr));
  }

  __half2 *sm_a = (__half2 *)(sm_buff + wg_shared_mem_size);
  for (int i = tid_in_wg; i < M * K / 2; i += 128) {
    sm_a[i] = gm_a[i];
  }

  __half2 *sm_b =
      (__half2 *)(sm_buff + wg_shared_mem_size + sizeof(__half2) * M * K / 2);

  for (int i = tid_in_wg; i < N * K / 2; i += 128) {
    sm_b[i] = gm_b[i];
  }

  for (int i = 0; i < 128; ++i) {
    RegD[i] = gm_c[tid_in_wg + 128 * i];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    // arrive
    uint64_t state = 0;
    asm volatile("{\n\t"
                 "mbarrier.arrive.shared.b64 %1, [%0];\n\t"
                 "}"
                 :
                 : "r"(bar_sm_addr), "l"(state));
  }

  uint32_t sm_a_addr = static_cast<uint32_t>(__cvta_generic_to_shared(sm_a));
  uint32_t sm_b_addr = static_cast<uint32_t>(__cvta_generic_to_shared(sm_b));

  GmmaDescriptor desc_a, desc_b;
  desc_a.layout_type_ = sm_layout;
  desc_b.layout_type_ = sm_layout;

  desc_a.start_address_ = sm_a_addr >> 4;
  desc_b.start_address_ = sm_b_addr >> 4;

  desc_a.base_offset_ = 0;
  desc_b.base_offset_ = 0;

  desc_a.leading_byte_offset_ = (8 * 8 * sizeof(__half)) >> 4;
  desc_b.leading_byte_offset_ = (8 * 8 * sizeof(__half)) >> 4;

  desc_a.stride_byte_offset_ = (2 * 8 * 8 * sizeof(__half)) >> 4;
  desc_b.stride_byte_offset_ = (2 * 8 * 8 * sizeof(__half)) >> 4;

  for (uint64_t repeat_i = 0; repeat_i < repeat_time; ++repeat_i) {
    // barrier
    // Arbitrarily large timer value after which try-wait expires and re-tries.
    uint32_t ticks = 0x989680;
    int phase = warp_group_id;
    asm volatile("{\n\t"
                 ".reg .pred       P1; \n\t"
                 "LAB_WAIT: \n\t"
                 "mbarrier.try_wait.parity.shared.b64 P1, [%0], %1, %2; \n\t"
                 "@P1 bra.uni DONE; \n\t"
                 "bra.uni     LAB_WAIT; \n\t"
                 "DONE: \n\t"
                 "}"
                 :
                 : "r"(bar_sm_addr), "r"(phase), "r"(ticks));

    // issue GMMA
    asm volatile(
        "{\n"
        "wgmma.mma_async.sync.aligned.m64n256k16.f32.f16.f16 "
        "{%0,   %1,   %2,   %3,   %4,   %5,   %6,   %7,   "
        " %8,   %9,   %10,  %11,  %12,  %13,  %14,  %15,  "
        " %16,  %17,  %18,  %19,  %20,  %21,  %22,  %23,  "
        " %24,  %25,  %26,  %27,  %28,  %29,  %30,  %31,  "
        " %32,  %33,  %34,  %35,  %36,  %37,  %38,  %39,  "
        " %40,  %41,  %42,  %43,  %44,  %45,  %46,  %47,  "
        " %48,  %49,  %50,  %51,  %52,  %53,  %54,  %55,  "
        " %56,  %57,  %58,  %59,  %60,  %61,  %62,  %63,  "
        " %64,  %65,  %66,  %67,  %68,  %69,  %70,  %71,  "
        " %72,  %73,  %74,  %75,  %76,  %77,  %78,  %79,  "
        " %80,  %81,  %82,  %83,  %84,  %85,  %86,  %87,  "
        " %88,  %89,  %90,  %91,  %92,  %93,  %94,  %95,  "
        " %96,  %97,  %98,  %99,  %100, %101, %102, %103, "
        " %104, %105, %106, %107, %108, %109, %110, %111, "
        " %112, %113, %114, %115, %116, %117, %118, %119, "
        " %120, %121, %122, %123, %124, %125, %126, %127},"
        " %128,"
        " %129,"
        " 1,   1,  1,  0,  0;\n"
        "}\n"
        : "+f"(RegD[0]), "+f"(RegD[1]), "+f"(RegD[2]), "+f"(RegD[3]),
          "+f"(RegD[4]), "+f"(RegD[5]), "+f"(RegD[6]), "+f"(RegD[7]),
          "+f"(RegD[8]), "+f"(RegD[9]), "+f"(RegD[10]), "+f"(RegD[11]),
          "+f"(RegD[12]), "+f"(RegD[13]), "+f"(RegD[14]), "+f"(RegD[15]),
          "+f"(RegD[16]), "+f"(RegD[17]), "+f"(RegD[18]), "+f"(RegD[19]),
          "+f"(RegD[20]), "+f"(RegD[21]), "+f"(RegD[22]), "+f"(RegD[23]),
          "+f"(RegD[24]), "+f"(RegD[25]), "+f"(RegD[26]), "+f"(RegD[27]),
          "+f"(RegD[28]), "+f"(RegD[29]), "+f"(RegD[30]), "+f"(RegD[31]),
          "+f"(RegD[32]), "+f"(RegD[33]), "+f"(RegD[34]), "+f"(RegD[35]),
          "+f"(RegD[36]), "+f"(RegD[37]), "+f"(RegD[38]), "+f"(RegD[39]),
          "+f"(RegD[40]), "+f"(RegD[41]), "+f"(RegD[42]), "+f"(RegD[43]),
          "+f"(RegD[44]), "+f"(RegD[45]), "+f"(RegD[46]), "+f"(RegD[47]),
          "+f"(RegD[48]), "+f"(RegD[49]), "+f"(RegD[50]), "+f"(RegD[51]),
          "+f"(RegD[52]), "+f"(RegD[53]), "+f"(RegD[54]), "+f"(RegD[55]),
          "+f"(RegD[56]), "+f"(RegD[57]), "+f"(RegD[58]), "+f"(RegD[59]),
          "+f"(RegD[60]), "+f"(RegD[61]), "+f"(RegD[62]), "+f"(RegD[63]),
          "+f"(RegD[64]), "+f"(RegD[65]), "+f"(RegD[66]), "+f"(RegD[67]),
          "+f"(RegD[68]), "+f"(RegD[69]), "+f"(RegD[70]), "+f"(RegD[71]),
          "+f"(RegD[72]), "+f"(RegD[73]), "+f"(RegD[74]), "+f"(RegD[75]),
          "+f"(RegD[76]), "+f"(RegD[77]), "+f"(RegD[78]), "+f"(RegD[79]),
          "+f"(RegD[80]), "+f"(RegD[81]), "+f"(RegD[82]), "+f"(RegD[83]),
          "+f"(RegD[84]), "+f"(RegD[85]), "+f"(RegD[86]), "+f"(RegD[87]),
          "+f"(RegD[88]), "+f"(RegD[89]), "+f"(RegD[90]), "+f"(RegD[91]),
          "+f"(RegD[92]), "+f"(RegD[93]), "+f"(RegD[94]), "+f"(RegD[95]),
          "+f"(RegD[96]), "+f"(RegD[97]), "+f"(RegD[98]), "+f"(RegD[99]),
          "+f"(RegD[100]), "+f"(RegD[101]), "+f"(RegD[102]), "+f"(RegD[103]),
          "+f"(RegD[104]), "+f"(RegD[105]), "+f"(RegD[106]), "+f"(RegD[107]),
          "+f"(RegD[108]), "+f"(RegD[109]), "+f"(RegD[110]), "+f"(RegD[111]),
          "+f"(RegD[112]), "+f"(RegD[113]), "+f"(RegD[114]), "+f"(RegD[115]),
          "+f"(RegD[116]), "+f"(RegD[117]), "+f"(RegD[118]), "+f"(RegD[119]),
          "+f"(RegD[120]), "+f"(RegD[121]), "+f"(RegD[122]), "+f"(RegD[123]),
          "+f"(RegD[124]), "+f"(RegD[125]), "+f"(RegD[126]), "+f"(RegD[127])
        : "l"(desc_a.desc_), "l"(desc_b.desc_));

    if (tid_in_wg == 0) {
      // arrive
      uint64_t state = 0;
      asm volatile("{\n\t"
                   "mbarrier.arrive.shared.b64 %1, [%0];\n\t"
                   "}"
                   :
                   : "r"(bar_sm_addr), "l"(state));
    }
  }

  asm volatile("wgmma.wait_group.sync.aligned %0;\n" ::"n"(0) : "memory");

  for (int i = 0; i < 128; ++i) {
    gm_d[tid_in_wg + 128 * i] = RegD[i];
  }
}

int main(int argc, char **argv) {
  int mat_a_size = M * K;
  int mat_b_size = N * K;
  int mat_c_size = M * N;

  __half *mat_a_host = new __half[mat_a_size];
  __half *mat_b_host = new __half[mat_b_size];
  float *mat_c_host = new float[mat_c_size];
  float *mat_d_host = new float[mat_c_size];

  __half2 *mat_a_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_a_dev, mat_a_size * sizeof(__half)));

  __half2 *mat_b_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_b_dev, mat_b_size * sizeof(__half)));

  float *mat_c_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_c_dev, mat_c_size * sizeof(float)));

  float *mat_d_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_d_dev, mat_c_size * sizeof(float)));

  int dyn_shared_size =
      (mat_a_size * sizeof(__half) + mat_b_size * sizeof(__half)) * 2 +
      sizeof(uint64_t);

  hipFuncSetAttribute(reinterpret_cast<const void*>(wgmma_m64n256k16_throughput_fmix_pingpong_test),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       dyn_shared_size);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int sm_count = prop.multiProcessorCount;
  std::cout << "using SM count: " << sm_count
            << " dynamic shared size: " << dyn_shared_size << "bytes"
            << std::endl;

  float duration;
  hipEvent_t start_event, stop_event;

  gpuErrchk(hipEventCreate(&start_event));
  gpuErrchk(hipEventCreate(&stop_event));
  gpuErrchk(hipEventRecord(start_event, 0));

  int repeat_time = 1024000;
  wgmma_m64n256k16_throughput_fmix_pingpong_test<<<sm_count, 256,
                                                   dyn_shared_size>>>(
      mat_d_dev, mat_a_dev, mat_b_dev, mat_c_dev, 0, repeat_time);
  gpuErrchk(hipEventRecord(stop_event, 0));
  gpuErrchk(hipEventSynchronize(stop_event));
  gpuErrchk(hipEventElapsedTime(&duration, start_event, stop_event));

  double FLOPS = M * K * N * 2 * 2;
  FLOPS *= repeat_time;
  FLOPS *= sm_count;

  double FLOPS_T = FLOPS / duration / 1000 / 1000 / 1000;

  std::cout << "duration: " << duration << "ms " << FLOPS_T << "TFLOP/s"
            << std::endl;
}
