#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_bf16.h>
#include <fstream>
#include <iostream>
#include <random>
#include <string>

#include "gmma_desc.h"

// code from
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

static const int M = 64;
static const int N = 256;
static const int K = 16;

// I didn't check result since it is just a throughput test
__global__ void
wgmma_m64n256k16_throughput_fmix_test(float *gm_d, __hip_bfloat162 *gm_a, __hip_bfloat162 *gm_b,
                                      float *gm_c, uint8_t sm_layout,
                                      uint64_t repeat_time) {
  extern __shared__ char sm_buff[];
  float RegD[128];

  __hip_bfloat162 *sm_a = (__hip_bfloat162 *)sm_buff;
  for (int i = threadIdx.x; i < M * K / 2; i += blockDim.x) {
    sm_a[i] = gm_a[i];
  }

  __hip_bfloat162 *sm_b = (__hip_bfloat162 *)(sm_buff + sizeof(__hip_bfloat162) * M * K / 2);

  for (int i = threadIdx.x; i < N * K / 2; i += blockDim.x) {
    sm_b[i] = gm_b[i];
  }

  for (int i = 0; i < 128; ++i) {
    RegD[i] = gm_c[threadIdx.x + blockDim.x * i];
  }

  __syncthreads();

  uint32_t sm_a_addr = static_cast<uint32_t>(__cvta_generic_to_shared(sm_a));
  uint32_t sm_b_addr = static_cast<uint32_t>(__cvta_generic_to_shared(sm_b));

  GmmaDescriptor desc_a, desc_b;
  desc_a.layout_type_ = sm_layout;
  desc_b.layout_type_ = sm_layout;

  desc_a.start_address_ = sm_a_addr >> 4;
  desc_b.start_address_ = sm_b_addr >> 4;

  desc_a.base_offset_ = 0;
  desc_b.base_offset_ = 0;

  desc_a.leading_byte_offset_ = (8 * 8 * sizeof(__hip_bfloat16)) >> 4;
  desc_b.leading_byte_offset_ = (8 * 8 * sizeof(__hip_bfloat16)) >> 4;

  desc_a.stride_byte_offset_ = (2 * 8 * 8 * sizeof(__hip_bfloat16)) >> 4;
  desc_b.stride_byte_offset_ = (2 * 8 * 8 * sizeof(__hip_bfloat16)) >> 4;

  for (uint64_t repeat_i = 0; repeat_i < repeat_time; ++repeat_i) {

    asm volatile(
        "{\n"
        "wgmma.mma_async.sync.aligned.m64n256k16.f32.bf16.bf16 "
        "{%0,   %1,   %2,   %3,   %4,   %5,   %6,   %7,   "
        " %8,   %9,   %10,  %11,  %12,  %13,  %14,  %15,  "
        " %16,  %17,  %18,  %19,  %20,  %21,  %22,  %23,  "
        " %24,  %25,  %26,  %27,  %28,  %29,  %30,  %31,  "
        " %32,  %33,  %34,  %35,  %36,  %37,  %38,  %39,  "
        " %40,  %41,  %42,  %43,  %44,  %45,  %46,  %47,  "
        " %48,  %49,  %50,  %51,  %52,  %53,  %54,  %55,  "
        " %56,  %57,  %58,  %59,  %60,  %61,  %62,  %63,  "
        " %64,  %65,  %66,  %67,  %68,  %69,  %70,  %71,  "
        " %72,  %73,  %74,  %75,  %76,  %77,  %78,  %79,  "
        " %80,  %81,  %82,  %83,  %84,  %85,  %86,  %87,  "
        " %88,  %89,  %90,  %91,  %92,  %93,  %94,  %95,  "
        " %96,  %97,  %98,  %99,  %100, %101, %102, %103, "
        " %104, %105, %106, %107, %108, %109, %110, %111, "
        " %112, %113, %114, %115, %116, %117, %118, %119, "
        " %120, %121, %122, %123, %124, %125, %126, %127},"
        " %128,"
        " %129,"
        " 1,   1,  1,  0,  0;\n"
        "}\n"
        : "+f"(RegD[0]), "+f"(RegD[1]), "+f"(RegD[2]), "+f"(RegD[3]),
          "+f"(RegD[4]), "+f"(RegD[5]), "+f"(RegD[6]), "+f"(RegD[7]),
          "+f"(RegD[8]), "+f"(RegD[9]), "+f"(RegD[10]), "+f"(RegD[11]),
          "+f"(RegD[12]), "+f"(RegD[13]), "+f"(RegD[14]), "+f"(RegD[15]),
          "+f"(RegD[16]), "+f"(RegD[17]), "+f"(RegD[18]), "+f"(RegD[19]),
          "+f"(RegD[20]), "+f"(RegD[21]), "+f"(RegD[22]), "+f"(RegD[23]),
          "+f"(RegD[24]), "+f"(RegD[25]), "+f"(RegD[26]), "+f"(RegD[27]),
          "+f"(RegD[28]), "+f"(RegD[29]), "+f"(RegD[30]), "+f"(RegD[31]),
          "+f"(RegD[32]), "+f"(RegD[33]), "+f"(RegD[34]), "+f"(RegD[35]),
          "+f"(RegD[36]), "+f"(RegD[37]), "+f"(RegD[38]), "+f"(RegD[39]),
          "+f"(RegD[40]), "+f"(RegD[41]), "+f"(RegD[42]), "+f"(RegD[43]),
          "+f"(RegD[44]), "+f"(RegD[45]), "+f"(RegD[46]), "+f"(RegD[47]),
          "+f"(RegD[48]), "+f"(RegD[49]), "+f"(RegD[50]), "+f"(RegD[51]),
          "+f"(RegD[52]), "+f"(RegD[53]), "+f"(RegD[54]), "+f"(RegD[55]),
          "+f"(RegD[56]), "+f"(RegD[57]), "+f"(RegD[58]), "+f"(RegD[59]),
          "+f"(RegD[60]), "+f"(RegD[61]), "+f"(RegD[62]), "+f"(RegD[63]),
          "+f"(RegD[64]), "+f"(RegD[65]), "+f"(RegD[66]), "+f"(RegD[67]),
          "+f"(RegD[68]), "+f"(RegD[69]), "+f"(RegD[70]), "+f"(RegD[71]),
          "+f"(RegD[72]), "+f"(RegD[73]), "+f"(RegD[74]), "+f"(RegD[75]),
          "+f"(RegD[76]), "+f"(RegD[77]), "+f"(RegD[78]), "+f"(RegD[79]),
          "+f"(RegD[80]), "+f"(RegD[81]), "+f"(RegD[82]), "+f"(RegD[83]),
          "+f"(RegD[84]), "+f"(RegD[85]), "+f"(RegD[86]), "+f"(RegD[87]),
          "+f"(RegD[88]), "+f"(RegD[89]), "+f"(RegD[90]), "+f"(RegD[91]),
          "+f"(RegD[92]), "+f"(RegD[93]), "+f"(RegD[94]), "+f"(RegD[95]),
          "+f"(RegD[96]), "+f"(RegD[97]), "+f"(RegD[98]), "+f"(RegD[99]),
          "+f"(RegD[100]), "+f"(RegD[101]), "+f"(RegD[102]), "+f"(RegD[103]),
          "+f"(RegD[104]), "+f"(RegD[105]), "+f"(RegD[106]), "+f"(RegD[107]),
          "+f"(RegD[108]), "+f"(RegD[109]), "+f"(RegD[110]), "+f"(RegD[111]),
          "+f"(RegD[112]), "+f"(RegD[113]), "+f"(RegD[114]), "+f"(RegD[115]),
          "+f"(RegD[116]), "+f"(RegD[117]), "+f"(RegD[118]), "+f"(RegD[119]),
          "+f"(RegD[120]), "+f"(RegD[121]), "+f"(RegD[122]), "+f"(RegD[123]),
          "+f"(RegD[124]), "+f"(RegD[125]), "+f"(RegD[126]), "+f"(RegD[127])
        : "l"(desc_a.desc_), "l"(desc_b.desc_));
  }

  asm volatile("wgmma.wait_group.sync.aligned %0;\n" ::"n"(0) : "memory");

  for (int i = 0; i < 128; ++i) {
    gm_d[threadIdx.x + blockDim.x * i] = RegD[i];
  }
}

template <typename T> void init_data(T *data, int count, std::string method);

template <> void init_data(__hip_bfloat16 *data, int count, std::string method) {
  if (method == "uniform") {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);

    for (int i = 0; i < count; ++i) {
      data[i] = __float2bfloat16(dis(gen));
    }
  } else {
    for (int i = 0; i < count; ++i) {
      data[i] = __float2bfloat16(0);
    }
  }
}

template <> void init_data(float *data, int count, std::string method) {
  if (method == "uniform") {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);

    for (int i = 0; i < count; ++i) {
      data[i] = dis(gen);
    }
  } else {
    for (int i = 0; i < count; ++i) {
      data[i] = 0;
    }
  }
}

int main(int argc, char **argv) {
  if (argc < 2) {
    std::cout << " usage ./test  <init method>\n";
    return -1;
  }

  std::string init_method(argv[1]);

  int mat_a_size = M * K;
  int mat_b_size = N * K;
  int mat_c_size = M * N;

  __hip_bfloat16 *mat_a_host = new __hip_bfloat16[mat_a_size];
  init_data(mat_a_host, mat_a_size, init_method);

  __hip_bfloat16 *mat_b_host = new __hip_bfloat16[mat_b_size];
  init_data(mat_b_host, mat_b_size, init_method);

  float *mat_c_host = new float[mat_c_size];
  init_data(mat_c_host, mat_c_size, init_method);

  float *mat_d_host = new float[mat_c_size];
  init_data(mat_d_host, mat_c_size, init_method);

  __hip_bfloat162 *mat_a_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_a_dev, mat_a_size * sizeof(__hip_bfloat16)));
  gpuErrchk(hipMemcpy(mat_a_dev, mat_a_host, mat_a_size * sizeof(__hip_bfloat16),
                       hipMemcpyHostToDevice));

  __hip_bfloat162 *mat_b_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_b_dev, mat_b_size * sizeof(__hip_bfloat16)));
  gpuErrchk(hipMemcpy(mat_b_dev, mat_b_host, mat_b_size * sizeof(__hip_bfloat16),
                       hipMemcpyHostToDevice));

  float *mat_c_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_c_dev, mat_c_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_c_dev, mat_c_host, mat_c_size * sizeof(float),
                       hipMemcpyHostToDevice));

  float *mat_d_dev = nullptr;
  gpuErrchk(hipMalloc(&mat_d_dev, mat_c_size * sizeof(float)));
  gpuErrchk(hipMemcpy(mat_d_dev, mat_d_host, mat_c_size * sizeof(float),
                       hipMemcpyHostToDevice));

  int dyn_shared_size =
      mat_a_size * sizeof(__hip_bfloat16) + mat_b_size * sizeof(__hip_bfloat16);

  hipFuncSetAttribute(reinterpret_cast<const void*>(wgmma_m64n256k16_throughput_fmix_test),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       dyn_shared_size);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int sm_count = prop.multiProcessorCount;
  std::cout << "using SM count: " << sm_count
            << " dynamic shared size: " << dyn_shared_size << "bytes"
            << std::endl;

  int repeat_time = 102400;
  int warmup_times = 0;
  // warm up
  for (int i = 0; i < warmup_times; ++i) {
    wgmma_m64n256k16_throughput_fmix_test<<<sm_count, 128, dyn_shared_size>>>(
        mat_d_dev, mat_a_dev, mat_b_dev, mat_c_dev, 0, repeat_time);
  }

  while (1) {
    float duration;
    hipEvent_t start_event, stop_event;

    gpuErrchk(hipEventCreate(&start_event));
    gpuErrchk(hipEventCreate(&stop_event));
    gpuErrchk(hipEventRecord(start_event, 0));

    wgmma_m64n256k16_throughput_fmix_test<<<sm_count, 128, dyn_shared_size>>>(
        mat_d_dev, mat_a_dev, mat_b_dev, mat_c_dev, 0, repeat_time);
    gpuErrchk(hipEventRecord(stop_event, 0));
    gpuErrchk(hipEventSynchronize(stop_event));
    gpuErrchk(hipEventElapsedTime(&duration, start_event, stop_event));
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    double FLOPS = M * K * N * 2;
    FLOPS *= repeat_time;
    FLOPS *= sm_count;

    double FLOPS_T = FLOPS / duration / 1000 / 1000 / 1000;

    std::cout << "duration: " << duration << "ms " << FLOPS_T << "TFLOP/s"
              << std::endl;
  }
}
