
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdint>


inline __host__ __device__ float4 make_float4(float s)
{
    return make_float4(s, s, s, s);
}


inline __host__ __device__ float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w);
}
inline __host__ __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    a.w += b.w;
}

__global__ void l2_float4_test(float4* data, float4* output_data, int64_t data_size, int64_t repeat) {
  int64_t init_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int64_t idx = init_idx;

  constexpr int unroll_size = 8;

  float4 temp_data[unroll_size];

  for (int ui = 0; ui < unroll_size; ui++) {
    temp_data[ui] = make_float4(0);
  }


  int stride = blockDim.x;

  for (int64_t i = 0; i < repeat; i += unroll_size) {
    for (int ui = 0; ui < unroll_size; ui++) {
      temp_data[ui] = temp_data[ui] + data[idx];
      idx += stride;
      if (idx >= data_size) {
        idx = init_idx;
      }
    }
  }
  float4 all_data = make_float4(0);
  for (int ui = 0; ui < unroll_size; ui++) {
    all_data = all_data + temp_data[ui];
  }

  output_data[threadIdx.x + blockIdx.x * blockDim.x] = all_data;
}


void test_l2_bw_float4(int repeat) {
  int32_t l2_size = 20*1024*1024;
  float4* l2_dev = nullptr;
  float4* output_data = nullptr;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int sm_count = prop.multiProcessorCount;

  int thread_num = 1024;

  hipMalloc(&l2_dev, l2_size);
  hipMalloc(&output_data, sm_count * thread_num * sizeof(float4));

  int data_size = l2_size / sizeof(float4);

  float device_time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  l2_float4_test<<<sm_count, thread_num>>>(l2_dev, output_data, data_size, repeat);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&device_time, start, stop);

  double bw = double(sm_count * thread_num) * repeat * sizeof(float4) / double(device_time) / 1000 / 1000;

  printf("test_l2_bw_float4 test time %5.2f bandwidth %6.2fG\n", device_time, bw);
}


__global__ void l2_float_stride_test(float* data, float* output_data, int64_t thread_stride, int64_t data_size, int64_t repeat) {
  int64_t init_idx = threadIdx.x * thread_stride + blockIdx.x * blockDim.x * thread_stride;
  int64_t idx = init_idx;

  constexpr int unroll_size = 8;

  float temp_data[unroll_size];

  for (int ui = 0; ui < unroll_size; ui++) {
    temp_data[ui] = 0.0f;
  }


  int stride = blockDim.x * thread_stride;

  for (int64_t i = 0; i < repeat; i += unroll_size) {
    for (int ui = 0; ui < unroll_size; ui++) {
      temp_data[ui] = temp_data[ui] + data[idx];
      idx += stride;
      if (idx >= data_size) {
        idx = init_idx;
      }
    }
  }
  float all_data = 0.0f;
  for (int ui = 0; ui < unroll_size; ui++) {
    all_data = all_data + temp_data[ui];
  }

  output_data[threadIdx.x + blockIdx.x * blockDim.x] = all_data;
}

void test_l2_bw_stride(int repeat, int thread_stride_byte) {
  int32_t l2_size = 20*1024*1024;
  float* l2_dev = nullptr;
  float* output_data = nullptr;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int sm_count = prop.multiProcessorCount;

  int thread_num = 1024;

  hipMalloc(&l2_dev, l2_size);
  hipMalloc(&output_data, sm_count * thread_num * sizeof(float));

  int data_size = l2_size / sizeof(float);

  float device_time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  l2_float_stride_test<<<sm_count, thread_num>>>(l2_dev, output_data, thread_stride_byte/sizeof(float), data_size, repeat);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&device_time, start, stop);

  double bw = double(sm_count * thread_num) * repeat * 32 / double(device_time) / 1000 / 1000;

  printf("test_l2_bw_stride test time %5.2f stride %4dB bandwidth %6.2fG\n", device_time, thread_stride_byte, bw);
}



int main(int argc, char** argv) {
  test_l2_bw_float4(100000);
  test_l2_bw_stride(100000, 32);
  test_l2_bw_stride(100000, 64);
  test_l2_bw_stride(100000, 128);
}

