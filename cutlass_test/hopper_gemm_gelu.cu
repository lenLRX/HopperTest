/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Simple Hopper GEMM example using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example demonstrate a simple way to instantiate and run a TF32 GEMM using the new CUTLASS 3.0
    APIs on NVIDIA Hopper architecture. New features that will be showcased in this example are as follows:

    1. NVIDIA Hopper architecture introduces a new series of tensor core instructions (GMMA) 
    which are more efficient than the Ampere tensor core instructions.

    2. NVIDIA Hopper architecture includes new Tensor Memory Accelerator (TMA) unit to transfer large 
    blocks of data efficiently between global memory and shared memory. TMA also supports asynchronous
    copies between thread blocks in a cluster. Another advantage is that TMA can load in FP32 data and
    convert them implicitly to TF32.

    3. This example uses the Warp Specialized kernel design (see /media/docs/efficient_gemm.md for details).

    Examples:

      $ ./examples/48_hopper_warp_specialized_gemm/48_hopper_warp_specialized_gemm --m=2048 --n=2048 --k=2048
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"


#include "cutlass/epilogue/thread/linear_combination_gelu.h"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::half_t;                                          // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::half_t;                                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;                                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_128,_64>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_4,_4,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;       // Kernel to launch based on the default setting in the Collective Builder 

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAuto,
    cutlass::gemm::collective::KernelScheduleAuto
  >::CollectiveOp;

namespace cutlass::epilogue::collective {

template <
  class ArchTag,
  class OpClass,
  class TileShape_MNK,
  class ClusterShape_MNK,
  class EpilogueTileType,
  class ElementAccumulator,
  class ElementCompute,
  class ElementC,
  class GmemLayoutTagC,
  int AlignmentC,
  class ElementD,
  class GmemLayoutTagD,
  int AlignmentD,
  class Schedule,
  class Enable = void
>
struct CollectiveGELUBuilder {
  static_assert(cutlass::detail::dependent_false<ArchTag>,
      "Could not build a collective epilogue for given parameters.");
};




// Auto builder
template <
  class TileShape_MNK,
  class ClusterShape_MNK,
  class EpilogueTileType,
  class ElementAccumulator,
  class ElementCompute,
  class ElementC,
  class GmemLayoutTagC,
  int AlignmentC,
  class ElementD,
  class GmemLayoutTagD,
  int AlignmentD,
  class Schedule
>
struct CollectiveGELUBuilder<
    arch::Sm90,
    arch::OpClassTensorOp,
    TileShape_MNK,
    ClusterShape_MNK,
    EpilogueTileType,
    ElementAccumulator,
    ElementCompute,
    ElementC,
    GmemLayoutTagC,
    AlignmentC,
    ElementD,
    GmemLayoutTagD,
    AlignmentD,
    Schedule,
    cute::enable_if_t<cute::is_same_v<Schedule, EpilogueScheduleAuto>>> {

private:
  static constexpr bool IsTmaAligned = cutlass::gemm::collective::detail::is_aligned<
      ElementC, AlignmentC, ElementD, AlignmentD, cutlass::gemm::collective::detail::tma_alignment_bytes>();

  // Current TMA epilogues require sixteen-bit data types and epilogue tile M to be of size 64.
  // Only dispatch to the TMA builder if these requirements are satisfied.
  static constexpr bool IsSixteenBit = sizeof_bits<ElementC>::value == 16 && sizeof_bits<ElementD>::value == 16;
  static constexpr bool IsEpiTileM64 = size<0>(shape(TileShape_MNK{})) == 64;

  using _CollectiveBuilder = CollectiveGELUBuilder<
    arch::Sm90,
    arch::OpClassTensorOp,
    TileShape_MNK,
    ClusterShape_MNK,
    EpilogueTileType,
    ElementAccumulator,
    ElementCompute,
    ElementC,
    GmemLayoutTagC,
    AlignmentC,
    ElementD,
    GmemLayoutTagD,
    AlignmentD,
    cute::conditional_t<IsTmaAligned && IsSixteenBit && IsEpiTileM64,
      TmaWarpSpecialized, NoSmemWarpSpecialized>
  >;

public:
  using ThreadOp = typename _CollectiveBuilder::ThreadOp;
  using CollectiveOp = typename _CollectiveBuilder::CollectiveOp;
};


__device__ float my_fast_gelu(float z) {
  float k0 = float(0.7978845608028654);
  float k1 = float(0.044715);

  return float(cutlass::constants::half<float>() * z *
      (cutlass::constants::one<float>() + fast_tanh(k0 * z * (cutlass::constants::one<float>() + k1 * z * z))));
}


/// Applies an element wise operation to all elements within the fragment
/// and writes them out to destination storage.
template <
  class StrideC_,
  class StrideD_,
  class ThreadEpilogueOp_,
  class EpilogueSchedule_
>
class DefaultGELUEpilogue {
public:
  //
  // Type Aliases
  //
  using EpilogueSchedule = EpilogueSchedule_;
  
  // derived types of output thread level operator
  using ThreadEpilogueOp = ThreadEpilogueOp_;
  using ElementOutput = typename ThreadEpilogueOp::ElementOutput;
  using ElementAccumulator = typename ThreadEpilogueOp::ElementAccumulator;
  using ElementCompute = typename ThreadEpilogueOp::ElementCompute;
  using ElementScalar = ElementCompute;
  using ElementC = typename ThreadEpilogueOp::ElementC;
  using StrideC = StrideC_;
  using ElementD = typename ThreadEpilogueOp::ElementD;
  using StrideD = StrideD_;

  using GmemTiledCopyC = void;
  using GmemTiledCopyD = void;

  static const int kOutputAlignment = ThreadEpilogueOp::kCount;
  using AlignmentType = typename cute::uint_bit<sizeof_bits<ElementOutput>::value * kOutputAlignment>::type;

  static_assert(rank(StrideC{}) == 3, "StrideCD must be rank-3: [M, N, L]");
  static_assert(rank(StrideD{}) == 3, "StrideCD must be rank-3: [M, N, L]");

  struct SharedStorage { };

  // Host side epilgoue arguments
  struct Arguments {
    typename ThreadEpilogueOp::Params thread{};
    ElementC const* ptr_C = nullptr;
    StrideC dC{};
    ElementD* ptr_D = nullptr;
    StrideD dD{};
  };

  // Device side epilogue params
  using Params = Arguments;

  //
  // Methods
  //

  template <class ProblemShape>
  static constexpr Params
  to_underlying_arguments(
      [[maybe_unused]] ProblemShape const& _,
      Arguments const& args,
      [[maybe_unused]] void* workspace) {
    return args;
  }

  CUTLASS_HOST_DEVICE
  DefaultGELUEpilogue(Params const& params_)
      : params(params_), epilogue_op(params_.thread) { }

  CUTLASS_DEVICE
  bool
  is_source_needed() {
    return epilogue_op.is_source_needed();
  }

  template<
    class ProblemShapeMNKL,
    class BlockShapeMNK,
    class BlockCoordMNKL,
    class FrgEngine, class FrgLayout,
    class TiledMma,
    class ResidueMNK
  >
  CUTLASS_HOST_DEVICE void
  operator()(
      ProblemShapeMNKL problem_shape_mnkl,
      BlockShapeMNK blk_shape_MNK,
      BlockCoordMNKL blk_coord_mnkl,
      cute::Tensor<FrgEngine, FrgLayout> const& accumulators,
      TiledMma tiled_mma,
      ResidueMNK residue_mnk,
      int thread_idx,
      [[maybe_unused]] char* smem_buf)
  {
    using namespace cute;
    using X = Underscore;

    static_assert(rank(ProblemShapeMNKL{}) == 4, "ProblemShapeMNKL must be rank 4");
    static_assert(is_static<BlockShapeMNK>::value, "ThreadBlock tile shape must be static");
    static_assert(rank(BlockShapeMNK{}) == 3, "BlockShapeMNK must be rank 3");
    static_assert(rank(BlockCoordMNKL{}) == 4, "BlockCoordMNKL must be rank 3");

    // Separate out problem shape for convenience
    auto M = get<0>(problem_shape_mnkl);
    auto N = get<1>(problem_shape_mnkl);
    auto L = get<3>(problem_shape_mnkl);

    auto stride_c = detail::get_epilogue_stride<EpilogueSchedule>(params.dC);
    auto stride_d = detail::get_epilogue_stride<EpilogueSchedule>(params.dD);

    // Represent the full output tensor
    Tensor mC_mnl = make_tensor(make_gmem_ptr(params.ptr_C), make_shape(M,N,L), stride_c);                 // (m,n,l)
    Tensor mD_mnl = make_tensor(make_gmem_ptr(params.ptr_D), make_shape(M,N,L), stride_d);                 // (m,n,l)
    Tensor gC_mnl = local_tile(mC_mnl, blk_shape_MNK, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)
    Tensor gD_mnl = local_tile(mD_mnl, blk_shape_MNK, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)

    // Slice to get the tile this CTA is responsible for
    auto [m_coord, n_coord, k_coord, l_coord] = blk_coord_mnkl;
    Tensor gC = gC_mnl(_,_,m_coord,n_coord,l_coord);                                                 // (BLK_M,BLK_N)
    Tensor gD = gD_mnl(_,_,m_coord,n_coord,l_coord);                                                 // (BLK_M,BLK_N)

    // Partition source and destination tiles to match the accumulator partitioning
    auto thr_mma = tiled_mma.get_thread_slice(thread_idx);
    Tensor tCgD = thr_mma.partition_C(gD);                                       // (VEC,THR_M,THR_N)
    Tensor tCgC = thr_mma.partition_C(gC);                                       // (VEC,THR_M,THR_N)

    static_assert(is_static<FrgLayout>::value, "Accumulator layout must be static");
    CUTE_STATIC_ASSERT_V(size(tCgC) == size(tCgD),
        "Source and destination must have the same number of elements.");
    CUTE_STATIC_ASSERT_V(size(tCgD) == size(accumulators),
        "Accumulator count must have the same destination element count.");

    // Make an identity coordinate tensor for predicating our output MN tile
    auto cD = make_identity_tensor(make_shape(unwrap(shape<0>(gD)), unwrap(shape<1>(gD))));
    Tensor tCcD = thr_mma.partition_C(cD);

    // source is needed
    if (epilogue_op.is_source_needed()) {
      CUTLASS_PRAGMA_UNROLL
      for (int i = 0; i < size(accumulators); ++i) {
        if (elem_less(tCcD(i), make_coord(get<0>(residue_mnk), get<1>(residue_mnk)))) {
          tCgD(i) = epilogue_op(accumulators(i), tCgC(i));
        }
      }
    }
    // source is not needed, avoid load
    else {
      CUTLASS_PRAGMA_UNROLL
      for (int i = 0; i < size(accumulators); ++i) {
        if (elem_less(tCcD(i), make_coord(get<0>(residue_mnk), get<1>(residue_mnk)))) {
          tCgD(i) = my_fast_gelu(accumulators(i));
        }
      }
    }
  }

private:
  Params params;
  ThreadEpilogueOp epilogue_op;
};




// No-smem builder
template <
  class TileShape_MNK,
  class ClusterShape_MNK,
  class EpilogueTileType,
  class ElementAccumulator,
  class ElementCompute,
  class ElementC_,
  class GmemLayoutTagC_,
  int AlignmentC,
  class ElementD,
  class GmemLayoutTagD,
  int AlignmentD,
  class Schedule
>
struct CollectiveGELUBuilder<
    arch::Sm90,
    arch::OpClassTensorOp,
    TileShape_MNK,
    ClusterShape_MNK,
    EpilogueTileType,
    ElementAccumulator,
    ElementCompute,
    ElementC_,
    GmemLayoutTagC_,
    AlignmentC,
    ElementD,
    GmemLayoutTagD,
    AlignmentD,
    Schedule,
    cute::enable_if_t<cute::is_same_v<Schedule, NoSmemWarpSpecialized>>> {

  // Passing void C disables source load
  using ElementC = cute::conditional_t<cute::is_void_v<ElementC_>,
      ElementD, ElementC_>; // prevents cute breakages
  using GmemLayoutTagC = cute::conditional_t<cute::is_void_v<ElementC_>,
      GmemLayoutTagD, GmemLayoutTagC_>;
  static constexpr thread::ScaleType::Kind ScaleType = cute::is_void_v<ElementC_> ?
      thread::ScaleType::OnlyAlphaScaling : thread::ScaleType::Default;

  static constexpr int FragmentSize = 1;
  using ThreadOp = thread::LinearCombination<
    ElementD, FragmentSize, ElementAccumulator, ElementCompute,
    ScaleType, FloatRoundStyle::round_to_nearest, ElementC>;

  using CollectiveOp = cutlass::epilogue::collective::detail::Sm90TmaWarpSpecializedAdapter<
    cutlass::epilogue::collective::DefaultGELUEpilogue<
      cutlass::gemm::TagToStrideC_t<GmemLayoutTagC>,
      cutlass::gemm::TagToStrideC_t<GmemLayoutTagD>,
      ThreadOp,
      cutlass::gemm::EpilogueDefault>
    >;
  //static_assert(false, "good");
};

} // end NS

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveGELUBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC, AlignmentC,
    ElementC, LayoutC, AlignmentC,
    cutlass::epilogue::collective::EpilogueScheduleAuto
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;

  Options():
    help(false),
    m(5120), n(4096), k(4096),
    alpha(1.f), beta(0.f),
    iterations(1000)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "48_hopper_warp_specialized_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "48_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, Int<1>{}));
  stride_B = make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, Int<1>{}));
  stride_C = make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, Int<1>{}));
  stride_D = make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, Int<1>{}));

  block_A.reset(options.m * options.k);
  block_B.reset(options.k * options.n);
  block_C.reset(options.m * options.n);
  block_D.reset(options.m * options.n);
  block_ref_D.reset(options.m * options.n);

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k},
    {block_A.get(), stride_A, block_B.get(), stride_B},
    {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
  };

  return arguments;
}

bool verify(const Options &options) {
  cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({options.m, options.k}));
  cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({options.n, options.k}));
  cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({options.m, options.n}));
  cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({options.m, options.n}));

  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    {options.m, options.n, options.k},
    ElementAccumulator(options.alpha),
    ref_A,
    ref_B,
    ElementAccumulator(options.beta),
    ref_C,
    ref_D);

  // Wait for kernel to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;


  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
